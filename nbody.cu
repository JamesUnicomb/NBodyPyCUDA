
#include <hip/hip_runtime.h>
#include <math.h>
#define EPS2 0.000001

__global__ void update(float4 *pos, float3 *vel, float4 *pos_, float3 *vel_, int n, float timedelta)
{
    float3 acc;
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    for (int sub_id = 0; sub_id < n; sub_id ++)
    {
        float3 r;

        r.x = pos_[sub_id].x - pos_[id].x;
        r.y = pos_[sub_id].y - pos_[id].y;
        r.z = pos_[sub_id].z - pos_[id].z;

        float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;
        float distSixth = distSqr * distSqr * distSqr;
        float invDistCube = 1.0f/sqrtf(distSixth);

        float s = pos_[id].w * invDistCube;

        acc.x += r.x * s;
        acc.y += r.y * s;
        acc.z += r.z * s;
        
    }

    vel[id].x = vel_[id].x + timedelta * acc.x;
    vel[id].y = vel_[id].y + timedelta * acc.y;
    vel[id].z = vel_[id].z + timedelta * acc.z;

    pos[id].x = pos_[id].x + timedelta * vel[id].x;
    pos[id].y = pos_[id].y + timedelta * vel[id].y;
    pos[id].z = pos_[id].z + timedelta * vel[id].z;
    pos[id].w = pos_[id].w;
}